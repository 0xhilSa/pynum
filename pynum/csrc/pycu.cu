#include <python3.10/Python.h>
#include <hip/hip_runtime.h>

#define CUDA_ERROR(call)                                        \
  do{                                                           \
    hipError_t err = call;                                     \
    if (err != hipSuccess){                                    \
      PyErr_Format(PyExc_RuntimeError,                          \
          "CUDA_ERROR: %s at %s:%d",                            \
          hipGetErrorString(err), __FILE__, __LINE__);         \
      return nullptr;                                           \
    }                                                           \
  }while (0)



